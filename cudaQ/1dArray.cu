#include "hip/hip_runtime.h"
//------------------------------------------------------------------------------
// compile with: nvcc -arch=sm_35 name.cu
//------------------------------------------------------------------------------
#include <stdlib.h>
#include <stdio.h>
 //-----------------------------------------------------------------------------
__global__ void vecAdd(int *xd, float *Ag, float *Bg, float *Cg) {
  // this is a kernel, which state the computations the gpu shall do
  //int j = threadIdx.x;
  int j = blockIdx.x*blockDim.x + threadIdx.x;
  *(Cg+j) = *(Ag+j) + *(Bg+j) + (*xd);
}
//------------------------------------------------------------------------------
int main() {
int N;
float *A, *B, *C;  // pointers to float
float *Ag, *Bg, *Cg;
int j;
N = 4;
int *xd;  // pointers to int
int *xdg;
int *xdn;
size_t sz = N*sizeof(float);  // this is the type of data for allocation funcs
size_t szi = sizeof(int);

// allocates cpu memory
A = (float *)malloc(sz);
B = (float *)malloc(sz);
C = (float *)malloc(sz);
xd = (int *)malloc(szi);
*xd = N;
xdn = (int *)malloc(szi);

printf("serial calc \n");
for (j = 0; j < N; j++) {
  A[j] = (float)j;  B[j] = (float)j;
  *(C+j) = *(A+j) + *(B+j) + (*xd);
  printf("A= %f B= %f A+B+xd= %f \n", *(A+j), *(B+j), *(C+j));
}

// allocates gpu memory
hipMalloc(&xdg, szi);  // notice that the pointer to a pointer is sent to hipMalloc
hipMalloc(&Ag, sz);
hipMalloc(&Bg, sz);
hipMalloc(&Cg, sz);

// copy data from cpu's memory to gpu's memory
hipMemcpy(xdg, xd, szi, hipMemcpyHostToDevice);
hipMemcpy(Ag, A, sz, hipMemcpyHostToDevice);
hipMemcpy(Bg, B, sz, hipMemcpyHostToDevice);
//hipMemcpy(Cg, C, sz, hipMemcpyHostToDevice);

dim3 blocksPerGrid(N/2,1,1);
// defines the No. of SMs to be used, for each dimension
dim3 threadsPerBloch(2,1,1);
// defines the No. of cores per SM to be used, for each dimension
// and runs the kernel in the gpu
vecAdd<<<blocksPerGrid, threadsPerBloch>>>(xdg, Ag, Bg, Cg);
// to wait for the gpu calc to end
hipDeviceSynchronize();

// copy data from gpu's memory to cpu's memory
hipMemcpy(xdn, xdg, szi, hipMemcpyDeviceToHost);
hipMemcpy(A, Ag, sz, hipMemcpyDeviceToHost);
hipMemcpy(B, Bg, sz, hipMemcpyDeviceToHost);
hipMemcpy(C, Cg, sz, hipMemcpyDeviceToHost);

printf("parallel calc \n");
for(j = 0; j < N; j++){
  printf("A= %f B= %f A+B+xd= %f \n", *(A+j), *(B+j), *(C+j));
}

// free gpu memory
hipFree(Ag); hipFree(Bg); hipFree(Cg); hipFree(xdg);
// free cpu memory
free(A); free(B); free(C); free(xd); free(xdn);

return 0;
}
//------------------------------------------------------------------------------
